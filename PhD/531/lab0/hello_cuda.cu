#include <stdio.h>
#include <hip/hip_runtime.h>


#define check(err) __check(err, __LINE__)
__host__ void __check(hipError_t err, int line) {
    if (err) {
        fprintf(stderr, "%d:%s\n", line, hipGetErrorString(err));
        abort();
    }
}

__global__ void hello_kernel() {
    int dev;
    hipGetDevice(&dev);
    printf("Hello from cuda device %d\n", dev);
}

void hello_cuda(int rank) {
    int count;
    check(hipGetDeviceCount(&count));
    check(hipSetDevice(rank % count));
    hello_kernel<<<1, 1>>>();
    hipDeviceSynchronize();
}
