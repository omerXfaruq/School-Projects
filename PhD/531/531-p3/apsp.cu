
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <stdint.h>

#define INF 200
__device__ inline int calc_loc_index(int tid)
{
    return 4 * tid;
}
__device__ inline int calc_kth_index(int tid)
{
    return 4 * tid + 1;
}

__device__ inline int calc_block_addr(uint x, int y, int r, int b)
{
    if (x <= y)
        return (x * b * r + y) * b;

    return (y * b * r + x) * b;
}

__device__ inline uint8_t calc_loc_val(int x, int y, uint8_t *data, int b)
{
    return data[4 * (x * b + y)];
}

__device__ inline uint8_t calc_kth_val(uint x, int y, uint8_t *data, int b)
{
    if (x < y)
        return data[4 * (x * b + y) + 1];

    return data[4 * (y * b + x) + 1];
}

__device__ inline uint8_t calc_neigh_val1(uint x, int y, uint8_t *data, int b)
{
    return data[4 * (x * b + y) + 1];
}

__device__ inline uint8_t calc_neigh_val2(uint x, int y, uint8_t *data, int b)
{
    return data[4 * (x * b + y) + 2];
}

uint8_t host_calc_val(int x, int y, uint8_t *data, int b)
{
    if (x == y)
        return 0;
    if (x < y)
        return data[x * b + y];

    return data[y * b + x];
}

__global__ void gloyd1(uint8_t *gdata, int N, int B, int R, int mrank)
{
    extern __shared__ uint8_t sdata[];
    int tid = threadIdx.x;
    int block_start_address = mrank * B * (R * B + 1);
    int loc_x = tid / B;
    int loc_y = tid % B;
    int gi = block_start_address + loc_x * R * B + loc_y;
    int gi_x = gi / R / B;
    int gi_y = gi % (R * B);

    if (gi_x < gi_y)
        sdata[calc_kth_index(tid)] = gdata[gi];

    for (int k = 0; k < B; k++) // TODO; reduce number of steps for edge blocks
    {
        __syncthreads();
        uint16_t w;
        if (gi_x < gi_y)
            w = calc_kth_val(loc_x, k, sdata, B) + calc_kth_val(k, loc_y, sdata, B);
        
        __syncthreads();
        
        if (gi_x < gi_y)
            if (w < sdata[calc_kth_index(tid)])
                sdata[calc_kth_index(tid)] = w;

            
    }
    if (gi_x < gi_y)
        gdata[gi] = sdata[calc_kth_index(tid)];
}

__global__ void gloyd2(uint8_t *gdata, int N, int B, int R, int cur_rank)
{
    extern __shared__ uint8_t sdata[];
    int tid = threadIdx.x;
    int pivot_block_start_address = cur_rank * B * (R * B + 1);
    int loc_x = tid / B;
    int loc_y = tid % B;
    int my_block_start_address, gi, p_gi, p_gi_x, p_gi_y;
    bool upside = (int)blockIdx.x < cur_rank;
    if (upside) // Up side of the pivot
        my_block_start_address = pivot_block_start_address - (cur_rank - blockIdx.x) * R * B * B;
    else // Right side of the pivot
        my_block_start_address = pivot_block_start_address + (blockIdx.x - cur_rank + 1) * B;

    gi = my_block_start_address + loc_x * R * B + loc_y;

    p_gi = pivot_block_start_address + loc_x * R * B + loc_y;
    p_gi_x = p_gi / R / B;
    p_gi_y = p_gi % (R * B);

    sdata[calc_loc_index(tid)] = gdata[gi];

    if (p_gi_x < p_gi_y)
        sdata[calc_kth_index(tid)] = gdata[p_gi];

    for (int k = 0; k < B; k++) // TODO; reduce number of steps for edge blocks
    {
        __syncthreads();
        uint16_t w;
        if (upside)
            w = calc_loc_val(loc_x, k, sdata, B) + calc_kth_val(k, loc_y, sdata, B);
        else
            w = calc_kth_val(loc_x, k, sdata, B) + calc_loc_val(k, loc_y, sdata, B);

        __syncthreads();

        if (w < sdata[calc_loc_index(tid)])
            sdata[calc_loc_index(tid)] = w;
    }

    gdata[gi] = sdata[calc_loc_index(tid)];
}

__global__ void gloyd3(uint8_t *gdata, int N, int B, int R, int cur_rank)
{
    extern __shared__ uint8_t sdata[];
    int tid = threadIdx.x;

    int block_row = blockIdx.x / R;
    int block_col = blockIdx.x % R;

    if (block_row > block_col || block_row == cur_rank || block_col == cur_rank)
        return;

    int kth_block_start_address_1 = calc_block_addr(block_row, cur_rank, R, B);
    int kth_block_start_address_2 = calc_block_addr(cur_rank, block_col, R, B);
    int my_block_start_address = calc_block_addr(block_row, block_col, R, B);

    int loc_x = tid / B;
    int loc_y = tid % B;

    int gi = my_block_start_address + loc_x * R * B + loc_y;
    int gi_x = gi / R / B;
    int gi_y = gi % (R * B);

    int p_gi1 = kth_block_start_address_1 + loc_x * R * B + loc_y;
    int p_gix1 = p_gi1 / R / B;
    int p_giy1 = p_gi1 % (R * B);

    int p_gi2 = kth_block_start_address_2 + loc_x * R * B + loc_y;
    int p_gix2 = p_gi2 / R / B;
    int p_giy2 = p_gi2 % (R * B);

    if (gi_x < gi_y)
        sdata[calc_loc_index(tid)] = gdata[gi];

    if (p_gix1 < p_giy1)
        sdata[calc_kth_index(tid)] = gdata[p_gi1];

    if (p_gix2 < p_giy2)
        sdata[calc_kth_index(tid) + 1] = gdata[p_gi2];

    for (int k = 0; k < B; k++) // TODO; reduce number of steps for edge blocks
    {
        __syncthreads();
        uint16_t w;
        uint8_t n1, n2;
        if (gi_x < gi_y)
        {
            if (block_row > cur_rank)
                n1 = calc_neigh_val1(k, loc_x, sdata, B); // Transpose to upper triangle
            else
                n1 = calc_neigh_val1(loc_x, k, sdata, B);

            if (cur_rank > block_col)
                n2 = calc_neigh_val2(loc_y, k, sdata, B); // Transpose to upper triangle
            else
                n2 = calc_neigh_val2(k, loc_y, sdata, B);

            w = n1 + n2;
        }

        __syncthreads();

        if (gi_x < gi_y)
            if (w < sdata[calc_loc_index(tid)])
                sdata[calc_loc_index(tid)] = w;


    }

    if (gi_x < gi_y)
        gdata[gi] = sdata[calc_loc_index(tid)];
}

int main(int argc, char **argv)
{
    int B = atoi(argv[3]);
    int n, m;
    uint8_t *d, *dd;
    // input
    FILE *infile = fopen(argv[1], "r");
    fscanf(infile, "%d %d", &n, &m);
    int R = n / B + (n % B != 0);
    int memsize = sizeof(uint8_t) * R * B * R * B;
    d = (uint8_t *)malloc(memsize);
    for (int i = 0; i < R * B * R * B; ++i)
        d[i] = INF;
    int a, b, w;
    for (int i = 0; i < m; ++i)
    {
        fscanf(infile, "%d %d %d", &a, &b, &w);
        d[a * R * B + b] = w;
    }
    fclose(infile);

    hipMalloc(&dd, memsize);
    hipMemcpy(dd, d, memsize, hipMemcpyHostToDevice);

    printf("R: %d N: %d B: %d\n", R, n, B);

    for (int r = 0; r < R; r++)
    {
        gloyd1<<<1, B * B, B * B * 4>>>(dd, n, B, R, r);
        gloyd2<<<R - 1, B * B, B * B * 4>>>(dd, n, B, R, r);
        gloyd3<<<R * R, B * B, B * B * 4>>>(dd, n, B, R, r);
    }

    hipDeviceSynchronize();
    hipMemcpy(d, dd, memsize, hipMemcpyDeviceToHost);

    // output
    FILE *outfile = fopen(argv[2], "w");
    for (int i = 0; i < n; ++i)
    {
        for (int j = 0; j < n; ++j)
        {
            fprintf(outfile, "%d%s",
                    (i == j ? 0 : host_calc_val(i, j, d, R * B)),
                    (j == n - 1 ? " \n" : " "));
        }
    }
    free(d);
    hipFree(dd);
}
